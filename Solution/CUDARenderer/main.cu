#include "hip/hip_runtime.h"
﻿
#include <iostream>
#include <time.h>
#include <float.h>
#include <fstream>

#include <hiprand/hiprand_kernel.h>
#include <>

#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "plane.h"
#include "hittable.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"

__device__ vec3 ray_color(const ray& r, hittable** scene, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1, 1, 1);
    vec3 emition = vec3(0, 0, 0);

    for (int i = 0; i < 12; i++) {
        hit_record rec;

        if ((*scene)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;  

            // If the material didn't produce scattered ray then it is emmisive. Calculating color and exiting loop
            if (!rec.mat_ptr->scatter(r, rec, attenuation, scattered, local_rand_state)) {
                emition = rec.mat_ptr->emitted();
                return emition * cur_attenuation;
            }

            // Adjusting attenuation based on current material
            cur_attenuation *= attenuation;
            cur_ray = scattered;
        }
        else {
            // Ray went out of bounds of the scene. Calculating color based on environment properties
            return vec3(0.0, 0.0, 0.0); // Current scene requires pitch black environment but I would put blue gradient here if wanted to simulate sky
        }
    }
    // Ray haven't found any light source and got lost somewhere in the scene geometry
    return vec3(0.0, 0.0, 0.0);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera** cam, hittable** scene, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Aborting the threads outside of the image bounds
    if ((i >= max_x) || (j >= max_y)) return;

    // Initializing rand_state for the thread
    int pixel_index = j * max_x + i;
    hiprand_init(0, pixel_index, 0, &rand_state[pixel_index]);
    hiprandState local_rand_state = rand_state[pixel_index];

    vec3 col(0, 0, 0);

    // Shooting rays from random points on the pixel, averaging the result
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += ray_color(r, scene, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;

    // Storing gamma corrected color to the frame buffer
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

__global__ void create_scene(hittable** d_list, hittable** d_scene, camera** d_camera, int nx, int ny) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        
        // Materials
        auto material_grey = new lambertian(vec3(0.99, 0.99, 0.99));
        auto material_emissive = new diffuse_light(vec3(1, 1, 1));
        auto material_green = new lambertian(vec3(0.01, 0.8, 0.01));
        auto material_red = new lambertian(vec3(0.8, 0.01, 0.01));
        auto material_chrome = new metal(vec3(0.8, 0.8, 0.8), 0.0);

        int i = 0;
        
        // Light source
        d_list[i++] = new xz_plane(-7, 7, 1, 15, 7.999, material_emissive);

        // Walls of the room
        d_list[i++] = new xz_plane(-8, 8, 0, 16, 8, material_grey);
        d_list[i++] = new xz_plane(-8, 8, 0, 16, 0, material_grey);
        d_list[i++] = new yz_plane(0, 8, 0, 16, 8, material_green);
        d_list[i++] = new yz_plane(0, 8, 0, 16, -8, material_red);
        d_list[i++] = new xy_plane(-8, 8, 0, 8, 16, material_grey);

        // Cube
        d_list[i++] = new xz_plane(1, 5, 11, 15, 4, material_grey);
        d_list[i++] = new yz_plane(0, 4, 11, 15, 5, material_grey);
        d_list[i++] = new yz_plane(0, 4, 11, 15, 1, material_grey);
        d_list[i++] = new xy_plane(1, 5, 0, 4, 11, material_grey);
        d_list[i++] = new xy_plane(1, 5, 0, 4, 15, material_grey);

        // Spheres
        d_list[i++] = new sphere(vec3(-2, 1.5, 8), 1.5, material_grey);
        d_list[i++] = new sphere(vec3(1.5, 0.6, 5), 0.6, material_grey);
        d_list[i++] = new sphere(vec3(0, 0.6, 6), 0.3, material_chrome);
        
        *d_scene = new hittable_list(d_list, i);

        // Setting up the camera
        vec3 lookfrom(0, 1, 0);
        vec3 lookat(0, 1, 1);

        float dist_to_focus = (lookfrom - lookat).length(); // Not needed for current scene
        float aperture = 0.1;                               // Not needed for current scene
        *d_camera = new camera(lookfrom,
            lookat,
            vec3(0, 1, 0),
            50,
            float(nx) / float(ny),
            aperture,
            dist_to_focus);
    }
}

int main() {
    // Image size
    int nx = 1920;
    int ny = 1080;

    // Samples per pixel
    int ns = 1024;

    // Block size (5x5 is optimal for my GPU)
    int tx = 5;
    int ty = 5;

    std::cout << "Image size: " << nx << "x" << ny << std::endl;
    std::cout << "Samples: " << ns << std::endl;
    std::cout << "Blocks: " << tx << "x" << ty << std::endl;

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // Allocating frame buffer
    vec3* fb;
    hipMallocManaged((void**)&fb, fb_size);

    // Allocating random state
    hiprandState* d_rand_state;
    hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState));

    // Allocating space for scene props
    hittable** d_list;
    int num_hittables = 14;
    hipMalloc((void**)&d_list, num_hittables * sizeof(hittable*));
    hittable** d_scene;
    hipMalloc((void**)&d_scene, sizeof(hittable*));
    camera** d_camera;
    hipMalloc((void**)&d_camera, sizeof(camera*));
    hipDeviceSynchronize();

    // Fill out the scene
    create_scene<<<1, 1>>>(d_list, d_scene, d_camera, nx, ny);
    hipDeviceSynchronize();

    clock_t start, stop;
    start = clock();

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    // Launch calculation
    render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_scene, d_rand_state);
    hipDeviceSynchronize(); // Wait for GPU to finish

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cout << "Elapsed time: " << timer_seconds << " seconds\n";

    // Outputting image to ppm file
    std::ofstream output("image.ppm");
    output << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            output << ir << " " << ig << " " << ib << "\n";
        }
    }
    output.close();

    hipDeviceSynchronize();
    hipDeviceReset();
}